#include "hip/hip_runtime.h"
#ifndef CUDA_ADABOOST_CORE_OPERATIONS_IMPL_HPP
#define CUDA_ADABOOST_CORE_OPERATIONS_IMPL_HPP

#include<adaboost/utils/utils.hpp>
#include<adaboost/cuda/core/operations.hpp>
#include<adaboost/cuda/utils/cuda_wrappers.hpp>
#include<adaboost/cuda/core/cuda_data_structures_impl.hpp>
#include<adaboost/core/operations_impl.cpp>
#include<iostream>
#include<cmath>

namespace adaboost
{
    namespace cuda
    {
        namespace core
        {
            template <class data_type_vector>
            __global__ void fill_vector_kernel
            (data_type_vector* data, unsigned size, data_type_vector value)
            {
                unsigned index = threadIdx.x;
                unsigned stride = blockDim.x;
                for(unsigned i = index; i < size; i += stride)
                {
                    data[i] = value;
                }
            }

            template <class data_type_vector>
            void fill(const data_type_vector value, const VectorGPU<data_type_vector>& vec, unsigned block_size)
            {
                bool gpu=true;
                if(block_size == 0)
                {
                    adaboost::core::fill(value, vec);
                }
                else
                {
                    fill_vector_kernel<data_type_vector>
                    <<<
                    (vec.get_size(gpu) + block_size - 1)/block_size, block_size
                    >>>
                    (vec.get_data_pointer(gpu), vec.get_size(gpu), value);
                }
            }

            template <class data_type_matrix>
            __global__ void fill_matrix_kernel
            (data_type_matrix* data, unsigned cols, data_type_matrix value)
            {
                unsigned row = blockDim.y*blockIdx.y + threadIdx.y;
                unsigned col = blockDim.x*blockIdx.x + threadIdx.x;
                data[row*cols + col] = value;
            }

            template <class data_type_matrix>
            void fill(const data_type_matrix value, const MatrixGPU<data_type_matrix>& mat, unsigned block_size_x, unsigned block_size_y)
            {
                bool gpu=true;
                if(block_size_x == 0 || block_size_y == 0)
                {
                    adaboost::core::fill(value, mat);
                }
                else
                {
                    dim3 gridDim((mat.get_cols(gpu) + block_size_x - 1)/block_size_x, (mat.get_rows(gpu) + block_size_y - 1)/block_size_y);
                    dim3 blockDim(block_size_x, block_size_y);
                    fill_matrix_kernel<data_type_matrix>
                    <<<
                    gridDim, blockDim
                    >>>
                    (mat.get_data_pointer(gpu), mat.get_rows(gpu), value);
                }
            }

        template <class data_type_vector>
            __global__
            void product_kernel
            (data_type_vector* v1, data_type_vector* v2, data_type_vector* v3,
            unsigned size)
            {
                __shared__ data_type_vector cache[MAX_BLOCK_SIZE];
                data_type_vector temp = 0;
                unsigned thread_i = threadIdx.x + blockDim.x*blockIdx.x;
                unsigned cache_i = threadIdx.x;
                while(thread_i < size)
                {
                    temp += v1[thread_i]*v2[thread_i];
                    thread_i = blockDim.x*gridDim.x;
                }
                cache[cache_i] = temp;
                __syncthreads();

                unsigned i = blockDim.x/2;
                while(i != 0)
                {
                    if(cache_i < i)
                    {
                        cache[cache_i] += cache[cache_i + i];
                    }
                    __syncthreads();
                    i /= 2;
                }

                if(cache_i == 0)
                    v3[blockIdx.x] = cache[0];
            }

            template <class data_type_vector>
            void product_gpu(const VectorGPU<data_type_vector>& vec1,
                             const VectorGPU<data_type_vector>& vec2,
                             data_type_vector& result,
                             unsigned block_size)
            {
                if(block_size == 0)
                {
                     adaboost::core::product(vec1, vec2, result);
                }
                else
                {
                    adaboost::utils::check(vec1.get_size() == vec2.get_size(),
                                           "Size of vectors don't match.");
                    adaboost::utils::check(block_size > 0,
                    "Size of the block should be a positive multiple of 32.");
                    unsigned num_blocks = (vec1.get_size() + block_size - 1)/block_size;
                    VectorGPU<data_type_vector> temp_result(num_blocks);
                    product_kernel
                    <<<
                    num_blocks,
                    block_size
                    >>>(vec1.get_data_pointer(), vec2.get_data_pointer(),
                        temp_result.get_data_pointer(), vec1.get_size());
                    temp_result.copy_to_host();
                    result = 0;
                    for(unsigned i = 0; i < num_blocks; i++)
                    {
                        result += temp_result.at(i);
                    }
                }
            }
            
            template <class data_type_matrix>
            __device__
            data_type_matrix get_element(
            data_type_matrix* mat,
            unsigned row,
            unsigned col,
            unsigned stride)
            {
                return mat[row*stride+col];
            }

            template <class data_type_matrix>
            __device__
            void set_element(
            data_type_matrix* mat,
            unsigned row,
            unsigned col,
            data_type_matrix value,
            unsigned stride)
            {
                mat[row*stride+col] = value;
            }

            template <class data_type_matrix>
            __device__
            data_type_matrix* get_sub_matrix(
            data_type_matrix* mat,
            unsigned block_row,
            unsigned block_col,
            unsigned stride)
            {
                data_type_matrix* mat_sub =
                new data_type_matrix[BLOCK_SIZE*BLOCK_SIZE];
                mat_sub = &mat[stride*BLOCK_SIZE*block_row+BLOCK_SIZE*block_col];
                return mat_sub;
            }

            template <class data_type_matrix>
            __global__
            void multiply_kernel(
            data_type_matrix* mat1,
            data_type_matrix* mat2,
            data_type_matrix* result,
            unsigned mat1_cols,
			unsigned mat1_rows,
            unsigned mat2_cols,
			unsigned mat2_rows,
            unsigned result_cols,
			unsigned result_rows)
            {
                unsigned block_row = blockIdx.y;
                unsigned block_col = blockIdx.x;
                data_type_matrix* result_sub = get_sub_matrix(result, block_row,
                                                              block_col, result_cols);

                unsigned row = threadIdx.y;
                unsigned col = threadIdx.x;
               
				__shared__ data_type_matrix mat1_shared[BLOCK_SIZE][BLOCK_SIZE];
                __shared__ data_type_matrix mat2_shared[BLOCK_SIZE][BLOCK_SIZE];
                data_type_matrix cvalue = 0.0;

                for(unsigned m = 0; m < (mat1_cols + BLOCK_SIZE - 1)/BLOCK_SIZE; m++)
                {
                    data_type_matrix* mat1_sub = get_sub_matrix(mat1, block_row,
                                                                m, mat1_cols);
                    data_type_matrix* mat2_sub = get_sub_matrix(mat2, m,
                                                                block_col, mat2_cols);

                    
					if (m*BLOCK_SIZE + col < mat1_cols && (block_row*BLOCK_SIZE+ row) < mat1_rows)
	                    mat1_shared[row][col] = get_element(mat1_sub, row, col, mat1_cols);
					else
						mat1_shared[row][col]=0;
					
					if (m*BLOCK_SIZE + row < mat2_rows && (block_col*BLOCK_SIZE+col) < mat2_cols)
    	                mat2_shared[row][col] = get_element(mat2_sub, row, col, mat2_cols);
					else
						mat2_shared[row][col]=0;

                    __syncthreads();

                    for(unsigned e = 0; e < BLOCK_SIZE; e++)
                    {
                        cvalue += mat1_shared[row][e] * mat2_shared[e][col];
                    }

                    __syncthreads();

                }
				if(block_row*BLOCK_SIZE+ row<result_rows && block_col*BLOCK_SIZE+col<result_cols)
                    set_element(result_sub, row, col, cvalue, result_cols);

            }

            template <class data_type_matrix>
            void multiply_gpu(const MatrixGPU<data_type_matrix>& mat1,
                              const MatrixGPU<data_type_matrix>& mat2,
                              MatrixGPU<data_type_matrix>& result)
            {
                adaboost::utils::check(mat1.get_cols() == mat2.get_rows(),
                                       "Order of matrices don't match.");
                dim3 gridDim((mat2.get_cols() + BLOCK_SIZE)/BLOCK_SIZE,
                             (mat1.get_rows() + BLOCK_SIZE)/BLOCK_SIZE);
                dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
                multiply_kernel
                <<<gridDim, blockDim>>>
                (mat1.get_data_pointer(),
                 mat2.get_data_pointer(),
                 result.get_data_pointer(),
                 mat1.get_cols(),
				 mat1.get_rows(),
                 mat2.get_cols(),
				 mat2.get_rows(),
                 result.get_cols(),
				 result.get_rows());
            }
            #include "../templates/instantiated_templates_cuda_operations.hpp"

        } //namespace cuda
    } //namespace core
} //namespace adaboost
#endif
